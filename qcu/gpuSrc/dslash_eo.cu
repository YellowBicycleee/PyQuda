#include "hip/hip_runtime.h"
#include "qcu.h"
#include <cstdio>
#include <time.h>
#include <cmath>
#include <assert.h>
#define NC 3
#define ND 4
#define NS 4
#define BLOCK_SIZE 128

#define checkCudaErrors(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "checkCudaErrors() API error = %04d \"%s\" from file <%s>, line %i.\n", \
                err, hipGetErrorString(err), __FILE__, __LINE__); \
                exit(-1); \
        }\
    }

#define getVecAddr(origin, x, y, z, t, Lx, Ly, Lz, Lt)  \
    ((origin) + ((((t) * (Lz) + (z)) * (Ly) + (y))*(Lx) + (x)) * NS * NC)   // 9 times
#define getGaugeAddr(origin, direction, x, y, z, t, Lx, Ly, Lz, Lt, even_odd) \
    ((origin) + (direction) * (Lt) * (Lz) * (Ly) * (Lx) * 2 * NC * NC + (even_odd) * ((Lt) * (Lz) * (Ly) * (Lx)) + ((((t) * (Lz) + (z)) * (Ly) + (y))*(Lx) + (x)) * NC * NC)


class Complex {
private:
    double real_;
    double imag_;
public:
    __device__ __host__
    Complex(double real, double imag) : real_(real), imag_(imag) { }
    __device__ __host__
    Complex() : real_(0), imag_(0) {}
    __device__ __host__
    Complex(const Complex& complex) : real_(complex.real_), imag_(complex.imag_){}
    __device__ __host__
    double norm2() {
        return sqrt(real_ * real_ + imag_ * imag_);
    }
    __device__ __host__
    void setImag(double imag) { imag_ = imag; }
    __device__ __host__
    void setReal(double real) { real_ = real; }
    __device__ __host__
    double real() const { return real_; }
    __device__ __host__
    double imag() const { return imag_; }

    __device__ __host__
    Complex& operator= (const Complex& complex) {
        real_ = complex.real_;
        imag_ = complex.imag_;
        return *this;
    }
    __device__ __host__
    Complex& operator= (double rhs) {
        real_ = rhs;
        imag_ = 0;
        return *this;
    }
    __device__ __host__
    Complex operator+(const Complex& complex) const {
        return Complex(real_+complex.real_, imag_+complex.imag_);
    }
    __device__ __host__
    Complex operator-(const Complex& complex) const {
        return Complex(real_-complex.real_, imag_-complex.imag_);
    }
    __device__ __host__
    Complex operator-() const{
        return Complex(-real_, -imag_);
    }
    __device__ __host__
    Complex operator*(const Complex& rhs) const {
        return Complex(real_ * rhs.real_ - imag_ * rhs.imag_, real_ * rhs.imag_ + imag_ * rhs.real_);
    }
    __device__ __host__
    Complex& operator*=(const Complex& rhs) {
        real_ = real_ * rhs.real_ - imag_ * rhs.imag_;
        imag_ = real_ * rhs.imag_ + imag_ * rhs.real_;
        return *this;
    }
    __device__ __host__
    Complex operator/ (const double& rhs) {
        return Complex(real_/rhs, imag_/rhs);
    }

    __device__ __host__
    Complex& operator+=(const Complex& rhs) {
        real_ += rhs.real_;
        imag_ += rhs.imag_;
        return *this;
    }

    __device__ __host__
    Complex& operator-=(const Complex& rhs) {
        real_ -= rhs.real_;
        imag_ -= rhs.imag_;
        return *this;
    }

    __device__ __host__
    Complex& clear2Zero() {
        real_ = 0;
        imag_ = 0;
        return *this;
    }
    __device__ __host__
    Complex conj() {
        return Complex(real_, -imag_);
    }
    __device__ __host__
    bool operator==(const Complex& rhs) {
        return real_ == rhs.real_ && imag_ == rhs.imag_;
    }
    __device__ __host__
    bool operator!=(const Complex& rhs) {
        return real_ != rhs.real_ || imag_ != rhs.imag_;
    }
    
};

// even_odd == 0---->even else odd
// transfer 
__global__
void gpuDslash(void* U_ptr, void* a_ptr, void* b_ptr, int Lx, int Ly, int Lz, int Lt, int even_odd) {
    // Lx >> 2;
    assert(even_odd == 0 || even_odd == 1);
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    // int odd_Lx = Lx;

    Lx >>= 1;
    int t = thread / (Lx * Ly * Lz);
    thread -= t * (Lx * Ly * Lz);
    int z = thread / (Lx * Ly);
    thread -= z * (Lx * Ly);
    int y = thread / Lx;
    int x = thread - y * Lx;
    // int old_Lx = Lx;
    // int sub_vol = Lt * Lz * Ly * Lx >> 2;


    int eo = (t+z+y) % 2;
    int pos_x;
    Complex *u;
    Complex *res;
    Complex *dest;
    Complex u_temp[NC * NC];            // for GPU
    Complex res_temp[NS * NC];          // for GPU
    Complex dest_temp[NS * NC];         // for GPU
    Complex u_last_line[NC];
    // double norm;

    Complex temp;
    for (int i = 0; i < NS*NC; i++) {
        dest_temp[i].clear2Zero();
    }
    dest = getVecAddr(static_cast<Complex*>(b_ptr), x, y, z, t, Lx, Ly, Lz, Lt);


    // \mu = 1
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 0, x, y, z, t, Lx, Ly, Lz, Lt, even_odd);
    // #pragma unroll
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj();// / norm;
    u_temp[7] = u_last_line[1].conj();// / norm;
    u_temp[8] = u_last_line[2].conj();// / norm;
    pos_x = (1 & ~(even_odd ^ eo)) * x +  (even_odd ^ eo) * ((x+1)%Lx);
    res = getVecAddr(static_cast<Complex*>(a_ptr), pos_x, y, z, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[3*NC+j] * Complex(0, 1)) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0,1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[2*NC+j] * Complex(0, 1)) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0,1);
        }
    }
    pos_x = (1 & ~(even_odd ^ eo)) * ((x-1+Lx) % Lx) + (even_odd ^ eo) * x;
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 0, pos_x, y, z, t, Lx, Ly, Lz, Lt, 1-even_odd);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // / norm;
    u_temp[7] = u_last_line[1].conj(); // / norm;
    u_temp[8] = u_last_line[2].conj(); // / norm;

    res = getVecAddr(static_cast<Complex*>(a_ptr), pos_x, y, z, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[3*NC+j] * Complex(0,1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0, -1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[2*NC+j] * Complex(0,1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0, -1);
        }
    }
    // \mu = 2
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 1, x, y, z, t, Lx, Ly, Lz, Lt, even_odd);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // / norm;
    u_temp[7] = u_last_line[1].conj(); // / norm;
    u_temp[8] = u_last_line[2].conj(); // / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, (y+1)%Ly, z, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[3*NC+j]) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[2*NC+j]) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += -temp;
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 1, x, (y+Ly-1)%Ly, z, t, Lx, Ly, Lz, Lt, 1-even_odd);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // / norm;
    u_temp[7] = u_last_line[1].conj(); // / norm;
    u_temp[8] = u_last_line[2].conj(); // / norm;

    res = getVecAddr(static_cast<Complex*>(a_ptr), x, (y+Ly-1)%Ly, z, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[3*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += -temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[2*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += temp;
        }
    }
    // \mu = 3
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 2, x, y, z, t, Lx, Ly, Lz, Lt, even_odd);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // norm;
    u_temp[7] = u_last_line[1].conj(); // norm;
    u_temp[8] = u_last_line[2].conj(); // norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, (z+1)%Lz, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[2*NC+j] * Complex(0, 1)) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0, 1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[3*NC+j] * Complex(0,1)) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0, -1);
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 2, x, y, (z+Lz-1)%Lz, t, Lx, Ly, Lz, Lt, 1-even_odd);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // norm;
    u_temp[7] = u_last_line[1].conj(); // norm;
    u_temp[8] = u_last_line[2].conj(); // norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, (z+Lz-1)%Lz, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[2*NC+j] * Complex(0, 1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0, -1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[3*NC+j] * Complex(0, 1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0, 1);
        }
    }
    // \mu = 4
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 3, x, y, z, t, Lx, Ly, Lz, Lt, even_odd);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // norm;
    u_temp[7] = u_last_line[1].conj(); // norm;
    u_temp[8] = u_last_line[2].conj(); // norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, z, (t+1)%Lt, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[2*NC+j]) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += -temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[3*NC+j]) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += -temp;
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 3, x, y, z, (t+Lt-1)%Lt, Lx, Ly, Lz, Lt, 1-even_odd);
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    // norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj(); // / norm;
    u_temp[7] = u_last_line[1].conj(); // / norm;
    u_temp[8] = u_last_line[2].conj(); // / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, z, (t+Lt-1)%Lt, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[2*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[3*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += temp;
        }
    }
    // end, copy result to dest
    for (int i = 0; i < NS * NC; i++) {
        dest[i] = dest_temp[i];
    }
}


void dslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param) {
    int even_odd = 0;   // waited to modify


    clock_t start, end;
    clock_t big_start, big_end;
    int Lx = param->lattice_size[0];
    int Ly = param->lattice_size[1];
    int Lz = param->lattice_size[2];
    int Lt = param->lattice_size[3];

    void* d_u;
    void* d_a;
    void* d_b;
    unsigned long u_size = ND * Lt * Lz * Ly * Lx * NC * NC * sizeof(Complex);
    unsigned long vec_size = Lt * Lz * Ly * Lx * NS * NC * sizeof(Complex) >> 1;
    int space = Lx * Ly * Lz * Lt >> 1;

    big_start = clock();
    checkCudaErrors(hipMalloc(&d_u, u_size));
    checkCudaErrors(hipMalloc(&d_a, vec_size));
    checkCudaErrors(hipMalloc(&d_b, vec_size));

    checkCudaErrors(hipMemcpy(d_u, gauge, u_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_a, fermion_in, vec_size, hipMemcpyHostToDevice));

    dim3 gridDim(space / BLOCK_SIZE);
    dim3 blockDim(BLOCK_SIZE);

    start = clock();
    // kernel function
    gpuDslash<<<gridDim, blockDim>>>(d_u, d_a, d_b, Lx, Ly, Lz, Lt, even_odd);
    hipError_t err = hipGetLastError();
    checkCudaErrors(err);
    // sync
    checkCudaErrors(hipDeviceSynchronize());
    end = clock();
    
    checkCudaErrors(hipMemcpy(fermion_out, d_b, vec_size, hipMemcpyDeviceToHost));
    // free memory
    checkCudaErrors(hipFree(d_u));
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    big_end = clock();
    printf("total time: (with malloc free memcpy) : %lf\n", (double)(big_end - big_start) / CLOCKS_PER_SEC);
    printf("total time: (without malloc free memcpy) : %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
}
